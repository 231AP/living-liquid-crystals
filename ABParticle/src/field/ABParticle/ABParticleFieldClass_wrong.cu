#include "hip/hip_runtime.h"
#ifndef ABPARTICLEFIELDCLASS_CU
#define ABPARTICLEFIELDCLASS_CU

#include <iostream> 
#include <vector>
#include <string>
#include <map>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "ABParticleFieldClass.h"
#include "ABParticleClassGPU.cu"


using namespace std;

// =============================================================
// Constructors
// -------------------------------------------------------------
ABParticleField::ABParticleField (Mesh* mesh_ptr_t, string name_t) {
    traits_host.mesh_ptr=mesh_ptr_t;
    traits_host.name=name_t;
    traits_host.priority=0;
    traits_host.boun_cond = "periodic";
    traits_host.init_cond = "sin";
    traits_host.location = "both";
    traits_host.expo_data = "on";    
    initPolarField ();
};


// -------------------------------------------------------------
ABParticleField::ABParticleField (Mesh* mesh_ptr_t, string name_t, int priority_t) {
    traits_host.mesh_ptr=mesh_ptr_t;
    traits_host.name=name_t;
    traits_host.priority=priority_t;
    traits_host.boun_cond = "periodic";
    traits_host.init_cond = "sin";
    traits_host.expo_data = "on";
    initPolarField ();
};


// -------------------------------------------------------------
ABParticleField::ABParticleField (Mesh* mesh_ptr_t, string name_t, int priority_t, string init_cond_t) {
    traits_host.mesh_ptr=mesh_ptr_t;
    traits_host.name=name_t;
    traits_host.priority=priority_t;
    traits_host.boun_cond = "periodic";
    traits_host.init_cond = init_cond_t;
    traits_host.expo_data = "on";
    initPolarField ();
};


// -------------------------------------------------------------
ABParticleField::ABParticleField (Mesh* mesh_ptr_t, string name_t, int priority_t, string init_cond_t, string boun_cond_t, string expo_data_t) {
    traits_host.mesh_ptr=mesh_ptr_t;
    traits_host.name=name_t;
    traits_host.priority=priority_t;
    traits_host.boun_cond = boun_cond_t;
    traits_host.init_cond = init_cond_t;
    traits_host.expo_data = expo_data_t;
    initPolarField ();
};


// -------------------------------------------------------------
void ABParticleField::initPolarField() {
    
};


// -------------------------------------------------------------
// =============================================================

void ExpoConf(const std::string& str_t) {
    std::ofstream ConfFile;
    //设置输出精度
    int PrecData = 8;

    // 文件名
    std::string ConfFileName = "conf_" + str_t + ".dat";
    ConfFile.open(ConfFileName.c_str());

    if (!ConfFile.is_open()) {
        std::cerr << "无法打开文件: " << ConfFileName << std::endl;
        return;
    }
    for (int idx = 0; idx < PM.particleNum; idx++) {
        // 使用固定格式和精度输出数据
        ConfFile << std::fixed << std::setprecision(PrecData)
            << pt.x[idx] << ' '
            << pt.y[idx];
        ConfFile << std::endl; // 换行
    }

    ConfFile.close();
}




// =============================================================


void MemFree() {
    // Free host memory
    delete[] pt.x;
    delete[] pt.y;

    delete[] pt.px;
    delete[] pt.py;

    delete[] pt.cellPx;
    delete[] pt.cellPy;

    delete[] pt.cellList;
    delete[] pt.cellOffsetsCL;
    delete[] pt.particleAroundId;
    delete[] pt.particleAroundFlagX;
    delete[] pt.particleAroundFlagY;
    delete[] pt.offsetsNL;
    delete[] pt.offsetsAL;
    delete[] pt.NeighborList;
    delete[] pt.NeighborListFlagX;
    delete[] pt.NeighborListFlagY;
    delete[] pt.fx;
    delete[] pt.fy;
    delete[] pt.x0ToUpdateHybridList;
    delete[] pt.y0ToUpdateHybridList;
    delete[] pt.state;

    // Free device memory
    hipFree(PT.x);
    hipFree(PT.y);

    hipFree(PT.px);
    hipFree(PT.py);
   


    hipFree(PT.cellX);
    hipFree(PT.cellY);

    hipFree(PT.cellPx);
    hipFree(PT.cellPy);




    hipFree(PT.cellList);
    hipFree(PT.cellOffsetsCL);
    hipFree(PT.particleAroundId);
    hipFree(PT.particleAroundFlagX);
    hipFree(PT.particleAroundFlagY);
    hipFree(PT.offsetsAL);
    hipFree(PT.offsetsNL);
    hipFree(PT.NeighborList);
    hipFree(PT.NeighborListFlagX);
    hipFree(PT.NeighborListFlagY);
    hipFree(PT.fx);
    hipFree(PT.fy);
    hipFree(PT.x0ToUpdateHybridList);
    hipFree(PT.y0ToUpdateHybridList);
    hipFree(PT.state);
}




// =============================================================
void getInput() {
    std::ifstream InputFile("input.dat");

    if (!InputFile.is_open()) {
        std::cerr << "Error opening input file!" << std::endl;
        return; // 退出函数
    }

    std::string line;
    int lineCount = 0;

    while (std::getline(InputFile, line)) {
        // 检查是否为注释行
        if (line.empty() || line.find('#') != std::string::npos) {
            continue; // 跳过空行和注释行
        }

        std::istringstream iss(line);
        switch (lineCount) {
        case 0: iss >> PM.boxX; break;
        case 1: iss >> PM.boxY; break;
        case 2: iss >> PM.cellSizeX; break;
        case 3: iss >> PM.cellSizeY; break;
        case 4: iss >> PM.cellNumX; break;
        case 5: iss >> PM.cellNumY; break;
        case 6: iss >> PM.rho; break;
        case 7: iss >> PM.maxParticlePerCell; break;
        case 8: iss >> PM.rd; break;
        case 9: iss >> PM.mask0; break;
        case 10: iss >> PM.mask1; break;
        case 11: iss >> PM.miniInstanceBetweenParticle; break;
        case 12: iss >> PM.r0; break;
        case 13: iss >> PM.epsilon; break;
        case 14: iss >> PM.kBT; break;
        case 15: iss >> PM.gammaValue; break;
        case 16: iss >> PM.rOutUpdateList; break;
        case 17: iss >> PM.particleNum; break;
        case 18: iss >> PM.tStart; break;
        case 19: iss >> PM.tStop; break;
        case 20: iss >> PM.tStep; break;
        case 21: iss >> PM.tExpo; break;
        case 22: iss >> PM.V0; break;
        case 23: iss >> PM.ABParticle; break;

        default: break; // 超过预期行数时不处理
        }
        lineCount++;
    }

    InputFile.close();
}







//=================================================================================





void MemAlloc() {
    // Allocate particle mem in host memory.
    pt.x = new real[PM.particleNum];
    pt.y = new real[PM.particleNum];


    pt.px = new real[PM.particleNum];
    pt.py = new real[PM.particleNum];



    pt.cellList = new int[PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell];
    pt.cellOffsetsCL = new int[PM.cellNumX * PM.cellNumY *2];
    pt.cellPx= new real[PM.cellNumX * PM.cellNumY * 2];
    pt.cellPy = new real[PM.cellNumX * PM.cellNumY * 2];


    pt.particleAroundId = new int[9 * PM.particleNum * PM.maxParticlePerCell];
    pt.particleAroundFlagX = new int[9 * PM.particleNum * PM.maxParticlePerCell];
    pt.particleAroundFlagY = new int[9 * PM.particleNum * PM.maxParticlePerCell];
    pt.offsetsNL = new int[PM.particleNum];
    pt.NeighborList = new int[PM.particleNum * PM.maxParticlePerCell];
    pt.NeighborListFlagX = new int[PM.particleNum];
    pt.NeighborListFlagY = new int[PM.particleNum];
    pt.fx = new real[PM.particleNum];
    pt.fy = new real[PM.particleNum];
    pt.x0ToUpdateHybridList = new real[PM.particleNum];
    pt.y0ToUpdateHybridList = new real[PM.particleNum];
    pt.state = new hiprandState[PM.particleNum];


    // Allocate memory of fields in device.
    hipMalloc((void**)&PT.x, PM.particleNum * sizeof(real));
    hipMalloc((void**)&PT.y, PM.particleNum * sizeof(real));



    hipMalloc((void**)&PT.px,PM.particleNum * sizeof(real));
    hipMalloc((void**)&PT.py,PM.particleNum * sizeof(real));



    hipMalloc((void**)&PT.cellX, PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.cellY, PM.particleNum * sizeof(int));


    hipMalloc((void**)&PT.cellList, PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell * sizeof(int));
    hipMalloc((void**)&PT.cellOffsetsCL, PM.cellNumX * PM.cellNumY   *2* sizeof(int));


    hipMalloc((void**)&PT.cellPx, PM.cellNumX * PM.cellNumY  * 2* sizeof(real));
    hipMalloc((void**)&PT.cellPy, PM.cellNumX * PM.cellNumY  * 2* sizeof(real));


    hipMalloc((void**)&PT.particleAroundId, 9 * PM.maxParticlePerCell * PM.particleNum * sizeof(int));  //这里以后可以把9改成作用力范围
    hipMalloc((void**)&PT.particleAroundFlagX, 9 * PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.particleAroundFlagY, 9 * PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.offsetsAL, PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.offsetsNL, PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.NeighborList, PM.particleNum * PM.maxParticlePerCell * sizeof(int));
    hipMalloc((void**)&PT.NeighborListFlagX, PM.particleNum * PM.maxParticlePerCell * sizeof(int));
    hipMalloc((void**)&PT.NeighborListFlagY, PM.maxParticlePerCell * PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.fx, PM.particleNum * sizeof(real));
    hipMalloc((void**)&PT.fy, PM.particleNum * sizeof(real));
    hipMalloc((void**)&PT.x0ToUpdateHybridList, PM.particleNum * sizeof(real));
    hipMalloc((void**)&PT.y0ToUpdateHybridList, PM.particleNum * sizeof(real));
    hipMalloc((void**)&PT.state, PM.particleNum * sizeof(hiprandState));
}

//=================================================================================







void printInput() {
    std::cout << "Box X: " << PM.boxX << std::endl;
    std::cout << "Box Y: " << PM.boxY << std::endl;
    std::cout << "Cell size X: " << PM.cellSizeX << std::endl;
    std::cout << "Cell size Y: " << PM.cellSizeY << std::endl;
    std::cout << "Cell num X: " << PM.cellNumX << std::endl;
    std::cout << "Cell num Y: " << PM.cellNumY << std::endl;
    std::cout << "Density: " << PM.rho << std::endl;
    std::cout << "Max particle per cell: " << PM.maxParticlePerCell << std::endl;
    std::cout << "Deadline distance: " << PM.rd << std::endl;
    std::cout << "Mask 0: " << PM.mask0 << std::endl;
    std::cout << "Mask 1: " << PM.mask1 << std::endl;
    std::cout << "Mini instance between particle: " << PM.miniInstanceBetweenParticle << std::endl;
    std::cout << "Equilibrium position: " << PM.r0 << std::endl;
    std::cout << "Epsilon: " << PM.epsilon << std::endl;
    std::cout << "kBT: " << PM.kBT << std::endl;
    std::cout << "Gamma value: " << PM.gammaValue << std::endl;
    std::cout << "Update list distance threshold: " << PM.rOutUpdateList << std::endl;
    std::cout << "Particle num: " << PM.particleNum << std::endl;
    std::cout << "Start time: " << PM.tStart << std::endl;
    std::cout << "Stop time: " << PM.tStop << std::endl;
    std::cout << "Time step: " << PM.tStep << std::endl;
    std::cout << "TExpo: " << PM.tExpo << std::endl;
    std::cout << "V0: " << PM.V0 << std::endl;
    std::cout << "ABParticle: " << PM.ABParticle << std::endl;
}

//=============================================================================

void Init_Coords(int flag, Particle pt, Parameter PM) {
    /*
    flag代表系统的初始化方式，flag=0代表均匀分布，flag=1代表随机分布
    当按照均匀分布时，需给定粒子密度，会同时按照初始粒子数目,初始系统的周期盒大小；
    当按照随机分布时，需给定粒子数目，随机生成粒子坐标
    */

    if (flag == 0) {
        //初始周期盒长度
        int N = PM.particleNum;
        real rho = PM.rho;
        real L = sqrt(N / rho);
        //考虑正方形盒子
        real xBox = L;
        real yBox = L;
        PM.boxX = xBox;
        PM.boxY = yBox;
        int initUcell = sqrt(N); //初始x,y,方向粒子数目
        real d_lattice = L / sqrt(N); //晶格间距
        //均匀分布 系统以原点为中心
        int n, nx, ny;
        n = 0;
        for (ny = 0;ny < initUcell; ny++) {
            for (nx = 0;nx < initUcell; nx++) {
                pt.x[n] = nx * d_lattice;
                pt.y[n] = ny * d_lattice;
                n++;
            }
        }
    }
    //随机分布 均匀分布的随机数生成器
    else if (flag == 1) {
        std::default_random_engine e;
        std::uniform_real_distribution<double> u(0.0, 1.0);
        e.seed(time(0));
        for (int n = 0; n < PM.particleNum; n++) {
            int flag = 0;
            pt.x[n] = u(e) * PM.boxX;
            pt.y[n] = u(e) * PM.boxY;

            pt.px[n] = u(e);
            pt.py[n] = u(e);



            while (1) {
                for (int m = 0; m < n; m++) {
                    // 计算两个粒子之间的距离，考虑周期性边界条件
                    float dx = fmod((pt.x[n] - pt.x[m] + PM.boxX), PM.boxX);
                    float dy = fmod((pt.y[n] - pt.y[m] + PM.boxY), PM.boxY);





                    // 若计算结果为负数，则调整到正值
                    if (dx > PM.boxX / 2) dx -= PM.boxX;
                    if (dy > PM.boxY / 2) dy -= PM.boxY;

                    // 计算距离的平方
                    float dist2 = dx * dx + dy * dy;

                    // 如果距离小于某个阈值（如 r0/2），则重新生成位置
                    if (dist2 < PM.r0 * PM.r0) {
                        flag = 1;
                        break;
                    }
                }

                // 如果找到距离太近的粒子，重新生成位置
                if (flag == 1) {
                    pt.x[n] = u(e) * PM.boxX;
                    pt.y[n] = u(e) * PM.boxY;
                    flag = 0;
                }
                else {
                    break;  // 如果所有的粒子距离都合适，退出循环
                }
            }

            //cout << u(e)<<"," << PM.boxX <<"," << pt.x[n] << endl;
        }
    }
    else if (flag == 2) {
        //计算粒子数
        int n = 0;
        int Ln = sqrt(PM.particleNum);
        //计算间距
        real dx = PM.boxX / (Ln - 1);
        real dy = PM.boxY / (Ln - 1);
        // 生成二维晶格的格点
        for (int i = 0; i < Ln; i++) {
            for (int j = 0; j < Ln; j++) {
                real x = j * dx; // 计算x坐标
                real y = i * dy; // 计算y坐标
                pt.x[n] = x;
                pt.y[n] = y;
                n++;
            }
        }
    }
}

//==============================================

void parameterInit() {
    PM.lenBit = 0;
    real boxToIntreal = PM.boxX / PM.miniInstanceBetweenParticle;
    while (++PM.lenBit) {//ignore boxX very small
        if (boxToIntreal < (1 << PM.lenBit)) break;
    }
    PM.lenBit++;
    PM.mask0 = (1 << PM.lenBit) + (1 << 2 * PM.lenBit + 1);
    int bitRd = ceil(log(PM.rd / PM.miniInstanceBetweenParticle) / log(2.0f));
    PM.mask1 = (((1 << (PM.lenBit - bitRd)) - 1) << bitRd) + (((1 << (PM.lenBit - bitRd)) - 1) << (bitRd + PM.lenBit + 1));
}

//上传=============================================================================================
void HostUpdataToDevice() {
    hipMemcpy(PT.x, pt.x, PM.particleNum * sizeof(real), hipMemcpyHostToDevice);
    hipMemcpy(PT.y, pt.y, PM.particleNum * sizeof(real), hipMemcpyHostToDevice);
}

//下载=============================================================================================
void DeviceUpdataToHost() {
    hipMemcpy(pt.x, PT.x, PM.particleNum * sizeof(real), hipMemcpyDeviceToHost);
    hipMemcpy(pt.y, PT.y, PM.particleNum * sizeof(real), hipMemcpyDeviceToHost);
}


//===========================================================================

void InitOffset() {
    hipMemset(PT.cellOffsetsCL, 0, sizeof(int) * PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell);
    hipMemset(PT.offsetsNL, 0, sizeof(int) * PM.particleNum);
    hipMemset(PT.offsetsAL, 0, sizeof(int) * PM.particleNum);
}

void listUpdate(Particle PT,Parameter PM) {
    InitOffset();
    getCellList << <PM.blockNum, PM.threadNum >> > (PT, PM);
    hipDeviceSynchronize();
    getAroundCellParticleId << <PM.blockNum, PM.threadNum >> > (PT, PM);
    hipDeviceSynchronize();
    saveXY0ToUpdateHybridList << <PM.blockNum, PM.threadNum >> > (PT, PM);
    hipDeviceSynchronize();
}



//===================================================================
void initBlockAndThreadNum() {
    PM.threadNum = 256;
    PM.blockNum = (PM.particleNum + PM.threadNum - 1) / PM.threadNum;
    printf("blockNum:%d,threadNum:%d\n", PM.blockNum, PM.threadNum);
}

//==========================================================================================================
// void showProgress(real tNow, real tStart, real tStop, clock_t clockNow, clock_t clockStart) {
//     real progress = (tNow - tStart) / (tStop - tStart);
//     real tUsed = double(clockNow - clockStart) / CLOCKS_PER_SEC;
//     real tUsePrediction = (tStop - tNow) * tUsed / (tNow - tStart);
//     printf("%.8f,%.8f  ", pt.x[0], pt.y[0]);
//     printf("  Progress:%.4f%%,%Prediction:%.1f\r", progress*100, tUsePrediction);
//     fflush(stdout);
// }

//=======================================================

//=========================================================================
void ABParticleField::forceAndPositionUpdate(Particle PT, Parameter PM,int i_field) {
 int Nx=gridNumber().x;
    int Ny=gridNumber().y;
    int Nbx=gridNumberBoun().x;
    int Nby=gridNumberBoun().y;
    double dx=gridSize().x;
    double dy=gridSize().y;



    getForce << <PM.blockNum, PM.threadNum>> > (PT, PM,(*ptr_vx).f[i_field],(*ptr_vy).f[i_field],(*ptr_Pxx).f[i_field],(*ptr_Pxy).f[i_field],(*ptr_Qxx).f[i_field],(*ptr_Qxy).f[i_field], Nx, Ny, Nbx, Nby);

    hipDeviceSynchronize();
    updatePosition << <PM.blockNum, PM.threadNum >> > (PT, PM);
    hipDeviceSynchronize();
}

//=========================================================================
void ABParticleField::iterate(Particle PT,Parameter PM,int i_field) {
    forceAndPositionUpdate(PT,PM,i_field);
    checkUpdate << <PM.blockNum, PM.threadNum >> > (PT, PM);
    hipMemcpyFromSymbol(&updateListFlagHost, HIP_SYMBOL(updateListFlag), sizeof(int));
    if (updateListFlagHost){
        listUpdate(PT, PM);
        updateListFlagHost = 0;
        hipMemcpyToSymbol(HIP_SYMBOL(updateListFlag), &updateListFlagHost, sizeof(int));
        
    };
    
    // CellPUpdata(PT,PM);

};

void ABParticleField::getConcentration(int i_field) {
    // Get velocity from vorticity field
    int Nx=gridNumber().x;
    int Ny=gridNumber().y;
    int Nbx=gridNumberBoun().x;
    int Nby=gridNumberBoun().y;
    double dx=gridSize().x;
    double dy=gridSize().y;
    

    if(i_field==0){
        iterate(PT,PM,i_field);
    // getLivingLCPxPyThetaGPU<<<Ny,Nx>>>((*ptr_flip).f[i_field],(*ptr_Pxx).f[i_field], f[i_field], (*ptr_px).f[i_field], (*ptr_py).f[i_field], (*ptr_theta).f[i_field], (*ptr_theta_old).f[i_field], Nx, Ny, Nbx, Nby);
    // getLivingLCFlipGPU<<<Ny,Nx>>>((*ptr_Omega).f[i_field],(*ptr_cplus).f[i_field],(*ptr_cminus).f[i_field],(*ptr_theta_old).f[i_field], (*ptr_theta).f[i_field],(*ptr_flip).f[i_field], Nx, Ny, Nbx, Nby);
    
    
    
    (*ptr_Pxx).applyBounCondPeriGPU((*ptr_Pxx).f[i_field]);
    // (*ptr_Concentration).applyBounCondPeriGPU((*ptr_Concentration).f[i_field]);
    (*ptr_Pxy).applyBounCondPeriGPU((*ptr_Pxy).f[i_field]);
    (*ptr_Qxx).applyBounCondPeriGPU((*ptr_Qxx).f[i_field]);
    (*ptr_Qxy).applyBounCondPeriGPU((*ptr_Qxy).f[i_field]);
     (*ptr_vx).applyBounCondPeriGPU((*ptr_vx).f[i_field]);
    (*ptr_vy).applyBounCondPeriGPU((*ptr_vy).f[i_field]);
    };
    
};


// =============================================================

#endif
